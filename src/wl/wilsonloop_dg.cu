#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <iostream>
#include <iomanip>



#include <timer.h>
#include <cuda_common.h>
#include <device_load_save.h>
#include <constants.h>
#include <matrixsun.h>
#include <gaugearray.h>
#include <index.h>
#include <device_PHB_OVR.h>
#include <reunitlink.h>
#include <staple.h>
#include <comm_mpi.h>
#include <exchange.h>
#include <texture_host.h>

#include <sharedmemtypes.h>

#include <tune.h>
#include <launch_kernel.cuh>


#include <cudaAtomic.h>

#include <hipcub/hipcub.hpp>



using namespace std;


namespace CULQCD{



template<class Real>
struct WLArg{
 const complex *gaugefield;
 const complex *fieldOp;
  Real *wloop;
  int radius;
  int Tmax;
  int mu;
  int opN;
  int iop;
};




#if __CUDA_ARCH__ >= 350
// Device has ldg
template<typename T>
__device__ __forceinline__ T Aldg(const T* ptr) {
    return __ldg(ptr);
}

#else
//Device does not, fall back.
template<typename T>
__device__ __forceinline__ T Aldg(const T* ptr) {
    return *ptr;
}
#endif





__constant__	int	 	DEV_Ops[2];
__constant__	int	 	DEV_OpComps[8];
__constant__	int	 	DEV_OpPos[8];


template<int blockSize, bool UseTex, class Real, ArrayType atype>
__global__ void kernel_WilsonLoop(WLArg<Real> arg){
  typedef hipcub::BlockReduce<Real, blockSize> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;  

  int id = INDEX1D();
	int x[4];
	Index_4D_NM(id, x);

	int idl = ( x[2] * param_Grid(1) + x[1] ) * param_Grid(0) + x[0]; //space index left
	x[arg.mu] = (x[arg.mu]+arg.radius)%param_Grid(arg.mu);
	int idr = ( x[2] * param_Grid(1) + x[1] ) * param_Grid(0) + x[0]; //space index right

	int tdirvolume = 3 * DEVPARAMS::Volume;
	int gfoffset = DEV_Ops[0] * DEVPARAMS::Volume;

	msun t0 = msun::identity();
	msun t1 = msun::identity();
	for(int it = 0; it <= arg.Tmax; it++){
		int idt = (x[3]+it)%param_Grid(3);
		idt *= DEVPARAMS::tstride;
        int idOp = 0;
        for(int comp = 0; comp < DEV_Ops[1]; comp++){
          for(int ii = 0; ii < DEV_OpComps[comp]; ii++){
          int iop = DEV_OpPos[comp] + ii;
          for(int iii = 0; iii < DEV_OpComps[comp]; iii++){
            int jop = DEV_OpPos[comp] + iii;

            msun linkb = msun::zero();
            if(id < DEVPARAMS::Volume) linkb = GAUGE_LOAD<false, atype, Real>( arg.fieldOp, id + iop * DEVPARAMS::Volume, gfoffset); //bottom space links
            Real w = 0.0;
		        if(id < DEVPARAMS::Volume){
			        msun linkt = GAUGE_LOAD_DAGGER<false, atype, Real>( arg.fieldOp, idl + idt + jop * DEVPARAMS::Volume, gfoffset); //top space links
			        w = (linkb * t1 * linkt * t0.dagger()).realtrace();
		        }
                int wloffset = it + idOp * (arg.Tmax+1); 
				Real aggregate = BlockReduce(temp_storage).Reduce(w, Summ<Real>());
				if (threadIdx.x == 0) CudaAtomicAdd(arg.wloop + wloffset, aggregate);
				__syncthreads();
                idOp++;
            }
        }
      }	      
		  if(id < DEVPARAMS::Volume && it < arg.Tmax){
			  t0 *= GAUGE_LOAD<UseTex, atype, Real>( arg.gaugefield, idl + idt + tdirvolume, DEVPARAMS::size);
			  t1 *= GAUGE_LOAD<UseTex, atype, Real>( arg.gaugefield, idr + idt + tdirvolume, DEVPARAMS::size);
		  } 
    }
}










template <bool UseTex, class Real, ArrayType atype> 
class WilsonLoop: Tunable{
private:
   WLArg<Real> arg;
   gauge array;
   int size;
   double timesec;
#ifdef TIMMINGS
    Timer WilsonLooptime;
#endif
	TuneParam tp;
	Real *wloop_tmp;
    size_t wloop_mem;
	Real *field_tmp;
    size_t field_mem;

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	tp = tuneLaunch(*this, getTuning(), getVerbosity());
  LAUNCH_KERNEL(kernel_WilsonLoop, tp, stream, arg, UseTex, Real, atype);
}
public:
   WilsonLoop(WLArg<Real> arg, gauge array): arg(arg), array(array){
	size = 1;
	for(int i=0;i<4;i++){
		size *= PARAMS::Grid[i];
	} 
	timesec = 0.0;  
}
  void SetIop(int iop){ arg.iop = iop;}
   ~WilsonLoop(){};
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    WilsonLooptime.start();
#endif
    apply(stream);
    CUDA_SAFE_DEVICE_SYNC();
    CUT_CHECK_ERROR("Kernel execution failed");
#ifdef TIMMINGS
	CUDA_SAFE_DEVICE_SYNC( );
    WilsonLooptime.stop();
    timesec = WilsonLooptime.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double time(){	return timesec;}
   void stat(){	COUT << "WilsonLoop:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size << ",prec="  << sizeof(Real);
    string tmp = "None";
    return TuneKey(vol.str().c_str(), typeid(*this).name(), tmp.c_str(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() { 
    int NidOp = 9;
    if(arg.opN == 1) NidOp = 1;
    wloop_mem = NidOp * (arg.Tmax+1) * sizeof(Real);
	wloop_tmp = (Real*) safe_malloc( wloop_mem );
  	CUDA_SAFE_CALL(hipMemcpy(wloop_tmp, arg.wloop, wloop_mem, hipMemcpyDeviceToHost));	
  }
  void postTune() {  
  	CUDA_SAFE_CALL(hipMemcpy(arg.wloop, wloop_tmp, wloop_mem, hipMemcpyHostToDevice));	
	host_free(wloop_tmp);
  }

};






template<bool UseTex, class Real>
void CalcWilsonLoop_dg(gauge array, gauge fieldOp, Real *wloop, int radius, int Tmax, int mu, int opN){
  Timer mtime;
  mtime.start(); 
  WLArg<Real> arg;
	arg.gaugefield = array.GetPtr();
	arg.fieldOp = fieldOp.GetPtr();
	arg.wloop = wloop;
	arg.radius = radius;
	arg.Tmax = Tmax;
	arg.mu = mu;
	arg.opN = opN;	

  
  if(array.Type() != SOA || fieldOp.Type() != SOA)
    errorCULQCD("Only defined for SOA arrays...\n");
  if(array.EvenOdd() == true || fieldOp.EvenOdd() == true)
    errorCULQCD("Not defined for EvenOdd arrays...\n");
    
  WilsonLoop<UseTex, Real, SOA> wl(arg, array);
  wl.Run();
  CUDA_SAFE_DEVICE_SYNC( );
  mtime.stop();
 if (getVerbosity() >= VERBOSE) COUT << "Time WilsonLoopF:  " <<  mtime.getElapsedTimeInSec() << " s"  << endl;
}



template<class Real>
void CalcWilsonLoop_dg(gauge array, gauge fieldOp, Real *wloop, int radius, int Tmax, int mu, int opN){
  if(PARAMS::UseTex){
    GAUGE_TEXTURE(array.GetPtr(), true);
    CalcWilsonLoop_dg<true, Real>(array, fieldOp, wloop, radius, Tmax, mu, opN);
  }
  else CalcWilsonLoop_dg<false, Real>(array, fieldOp, wloop, radius, Tmax, mu, opN);
}


template void CalcWilsonLoop_dg<double>(gauged array, gauged fieldOp, double *wloop, int radius, int Tmax, int mu, int opN);








}
