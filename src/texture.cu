#include "hip/hip_runtime.h"



#include <texture.h>


namespace CULQCD{
//////////////////////////////////////////////////////////////////////////////////////
/*
  TEXTURES
*/
//////////////////////////////////////////////////////////////////////////////////////
texture<float2, 1, hipReadModeElementType> tex_gauge_float;
texture<int4, 1, hipReadModeElementType> tex_gauge_double;

texture<float2, 1, hipReadModeElementType> tex_gx_float;
texture<int4, 1, hipReadModeElementType> tex_gx_double;

texture<float2, 1, hipReadModeElementType> tex_delta_float;
texture<int4, 1, hipReadModeElementType> tex_delta_double;

texture<float2, 1, hipReadModeElementType> tex_lambda_float;
texture<int4, 1, hipReadModeElementType> tex_lambda_double;



}