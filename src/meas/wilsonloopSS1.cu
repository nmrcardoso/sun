#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <iostream>
#include <iomanip>



#include <timer.h>
#include <cuda_common.h>
#include <device_load_save.h>
#include <constants.h>
#include <matrixsun.h>
#include <gaugearray.h>
#include <index.h>
#include <device_PHB_OVR.h>
#include <reunitlink.h>

#include <comm_mpi.h>
#include <exchange.h>
#include <texture_host.h>

#include <sharedmemtypes.h>

#include <tune.h>
#include <launch_kernel.cuh>


#include <cudaAtomic.h>

#include <hipcub/hipcub.hpp>


#include <reduce_block_1d.h>

using namespace std;


namespace CULQCD{


//#define __WILSON_LOOP_USE_CUB__



template<class Real>
struct WLArgR{
  complex *gaugefield;
  complex *gaugefield_nosmear;
  complex *WLsp;
  complex *res;
  int radius;
};





template<bool UseTex, class Real, ArrayType atype>
__global__ void kernel_WilsonLineSP1(WLArgR<Real> arg){
    int id = INDEX1D();

    if(id >= DEVPARAMS::Volume) return;

    for(int mu = 0; mu < 3; mu++){
        msun link = msun::identity();
        for(int radius = 0; radius < arg.radius; radius++){
            link *= GAUGE_LOAD<UseTex, atype, Real>( arg.gaugefield, Index_4D_Neig_NM(id, mu, radius) + mu * DEVPARAMS::Volume, DEVPARAMS::size);
	        GAUGE_SAVE<atype, Real>( arg.WLsp, link, id + mu * DEVPARAMS::Volume + radius * DEVPARAMS::Volume * 3, DEVPARAMS::Volume * 3 * arg.radius );

            }
    }
}




#ifdef __WILSON_LOOP_USE_CUB__
template<int blockSize, bool UseTex, class Real, ArrayType atype>
__global__ void kernel_WilsonLoopSSP1(WLArgR<Real> arg){
  typedef hipcub::BlockReduce<complex, blockSize> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
#else
template<bool UseTex, class Real, ArrayType atype>
__global__ void kernel_WilsonLoopSSP1(WLArgR<Real> arg){
#endif       

  int id = INDEX1D();
	

  for(int mu = 0; mu < 3; mu++)
  for(int nu = 0; nu < 3; nu++){
  	if(mu==nu) continue;
	int tdirvolume = nu * DEVPARAMS::Volume;
    
    for(int radius = 0; radius <= arg.radius; radius++){
        msun linkb = msun::identity();
        if(id < DEVPARAMS::Volume && radius > 0)
            linkb = GAUGE_LOAD<UseTex, atype, Real>( arg.WLsp, id + mu * DEVPARAMS::Volume + (radius-1) * DEVPARAMS::Volume * 3, DEVPARAMS::Volume * 3 * arg.radius);
        
	    msun t0 = msun::identity();
	    msun t1 = msun::identity();
	    for(int it = 0; it <= arg.radius; it++){

		    int idt = Index_4D_Neig_NM(id, nu, it);
		    msun linktop = msun::identity();
		    if(id < DEVPARAMS::Volume && radius > 0)
		        linktop = GAUGE_LOAD<UseTex, atype, Real>( arg.WLsp, idt + mu * DEVPARAMS::Volume + (radius-1) * DEVPARAMS::Volume * 3, DEVPARAMS::Volume * 3 * arg.radius); 
     
			  complex wl = complex::zero();
			  if(id < DEVPARAMS::Volume) wl = (linkb * t1 * linktop.dagger() * t0.dagger()).trace();
               
#ifdef __WILSON_LOOP_USE_CUB__
			  complex aggregate;
			  aggregate = BlockReduce(temp_storage).Reduce(wl, Summ<complex>());
			  if (threadIdx.x == 0) CudaAtomicAdd(arg.res + it + (arg.radius+1) * radius, aggregate);

#else
			  reduce_block_1d<complex>(arg.res + it + (arg.radius+1) * radius, wl);
#endif
			  __syncthreads();


		    if(id < DEVPARAMS::Volume && it < arg.radius){
			    t0 *= GAUGE_LOAD<UseTex, atype, Real>( arg.gaugefield_nosmear, idt + tdirvolume, DEVPARAMS::size);
			    t1 *= GAUGE_LOAD<UseTex, atype, Real>( arg.gaugefield_nosmear, Index_4D_Neig_NM(idt, mu, radius) + tdirvolume, DEVPARAMS::size);
		    } 

     	 }
    }
  }
}


template <bool UseTex, class Real, ArrayType atype> 
class WilsonLineSP1: Tunable{
private:
   WLArgR<Real> arg;
   gauge array;
   int size;
   double timesec;
#ifdef TIMMINGS
    Timer WilsonLinetime;
#endif
	TuneParam tp;

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
        tp = tuneLaunch(*this, getTuning(), getVerbosity());
        kernel_WilsonLineSP1<UseTex, Real, atype><<<tp.grid,tp.block, 0, stream>>>(arg);
}
public:
   WilsonLineSP1(WLArgR<Real> arg, gauge array): arg(arg), array(array){
	size = 1;
	for(int i=0;i<4;i++){
		size *= PARAMS::Grid[i];
	} 
	timesec = 0.0;  
}
   ~WilsonLineSP1(){};
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    WilsonLinetime.start();
#endif
    apply(stream);
    CUDA_SAFE_DEVICE_SYNC();
    CUT_CHECK_ERROR("Kernel execution failed");
#ifdef TIMMINGS
	CUDA_SAFE_DEVICE_SYNC( );
    WilsonLinetime.stop();
    timesec = WilsonLinetime.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const {	return 0;}
   long long bytes() const{return 0;}
   double time(){	return timesec;}
   void stat(){	COUT << "WilsonLineSP:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size << ",prec="  << sizeof(Real);
    string tmp = "None";
    return TuneKey(vol.str().c_str(), typeid(*this).name(), tmp.c_str(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() {  }
  void postTune() {    }

};






template <bool UseTex, class Real, ArrayType atype> 
class WilsonLoopSSSP1: Tunable{
private:
   WLArgR<Real> arg;
   gauge array;
   int size;
   double timesec;
#ifdef TIMMINGS
    Timer WilsonLoopSStime;

#endif
	TuneParam tp;

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.

   unsigned int minThreads() const { return size; }

   void apply(const hipStream_t &stream){
        tp = tuneLaunch(*this, getTuning(), getVerbosity());
        CUDA_SAFE_CALL(hipMemset(arg.res, 0, (arg.radius + 1) * (arg.radius+1) * sizeof(complex)));
#ifdef __WILSON_LOOP_USE_CUB__
        LAUNCH_KERNEL(kernel_WilsonLoopSSP1, tp, stream, arg, UseTex, Real, atype);
#else
		kernel_WilsonLoopSSP1<UseTex, Real, atype><<<tp.grid,tp.block, tp.block.x*sizeof(complex), stream>>>(arg);
#endif

}
public:
   WilsonLoopSSSP1(WLArgR<Real> arg, gauge array): arg(arg), array(array){
	size = 1;
	for(int i=0;i<4;i++){
		size *= PARAMS::Grid[i];
	} 
	timesec = 0.0;  
}
   ~WilsonLoopSSSP1(){};
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    WilsonLoopSStime.start();
#endif
    apply(stream);
    CUDA_SAFE_DEVICE_SYNC();
    CUT_CHECK_ERROR("Kernel execution failed");
#ifdef TIMMINGS
	CUDA_SAFE_DEVICE_SYNC( );
    WilsonLoopSStime.stop();
    timesec = WilsonLoopSStime.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const {	return 0;}
   long long bytes() const{return 0;}
   double time(){	return timesec;}
   void stat(){	COUT << "WilsonLoopSS:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size << ",prec="  << sizeof(Real);
    string tmp = "None";
    return TuneKey(vol.str().c_str(), typeid(*this).name(), tmp.c_str(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() {  }
  void postTune() {    }

};








template<bool UseTex, class Real>
void WilsonLoopSS1(gauge array, gauge array_nosmear, complex *res, int radius){
  Timer mtime;
  mtime.start(); 
  WLArgR<Real> arg;
	arg.gaugefield = array.GetPtr();
  	arg.gaugefield_nosmear = array_nosmear.GetPtr();
	arg.res = (complex*) dev_malloc((radius+1)*(radius+1)*sizeof(complex));
	arg.radius = radius;

	gauge WLsp(SOA, Device, PARAMS::Volume * 3 * radius, false);
    arg.WLsp = WLsp.GetPtr();
  
  if(array.Type() != SOA)
    errorCULQCD("Only defined for SOA arrays...\n");
  if(array.EvenOdd() == true)
    errorCULQCD("Not defined for EvenOdd arrays...\n");
  if(array_nosmear.Type() != SOA)
    errorCULQCD("Only defined for SOA arrays...\n");
  if(array_nosmear.EvenOdd() == true)
    errorCULQCD("Not defined for EvenOdd arrays...\n");



    WilsonLineSP1<UseTex, Real, SOA> wline(arg, array);
    WilsonLoopSSSP1<UseTex, Real, SOA> wlsp(arg, array);
    wline.Run();
    wline.stat();
    wlsp.Run();
    wlsp.stat();
    WLsp.Release();


  CUDA_SAFE_CALL(hipMemcpy(res, arg.res, (arg.radius + 1) * (arg.radius+1) * sizeof(complex), hipMemcpyDeviceToHost));
  dev_free(arg.res);
  for(int r = 0; r <= radius; r++)
  for(int it = 0; it <= radius; it++)
    res[it + r * (radius+1)] /= (Real)(PARAMS::Volume * 3 * NCOLORS);
  CUDA_SAFE_DEVICE_SYNC( );
  mtime.stop();
  COUT << "Time WilsonLoopSS:  " <<  mtime.getElapsedTimeInSec() << " s"  << endl;
}




template<class Real>
void WilsonLoopSS1(gauge array, gauge array_nosmear, complex *res, int radius){
  if(PARAMS::UseTex){
    GAUGE_TEXTURE(array.GetPtr(), true);
    WilsonLoopSS1<true, Real>(array, array_nosmear, res, radius);
  }
  else WilsonLoopSS1<false, Real>(array, array_nosmear, res, radius);
}


template void WilsonLoopSS1<float>(gauges array, gauges array_nosmear, complexs *res, int radius);
template void WilsonLoopSS1<double>(gauged array, gauged array_nosmear, complexd *res, int radius);





































































template<class Real>
struct WLArgRnoInt{
  complex *gaugefield;
  complex *gaugefield_nosmear;
  complex *WLsp;
  complex *res;
  int radius[3];
  int Tmax;
  int mu;
  int nu;
};





template<bool UseTex, class Real, ArrayType atype>
__global__ void kernel_WilsonLoopRnoInt(WLArgRnoInt<Real> arg){
          

	int id = INDEX1D();
	
	
    //if(id==0) printf("%d::%d::%d::\n",arg.radius[0],arg.radius[1],arg.radius[2]);
    
	

	int tdirvolume = arg.nu * DEVPARAMS::Volume;

    msun linkb = msun::identity();
	int ids = id;
	if(id < DEVPARAMS::Volume){
		for(int dir = 0; dir < 3; dir++){
			int newdir = dir;//( arg.mu + dir ) % 3;
			for(int radius = 0; radius < arg.radius[newdir]; radius++){
				linkb *= GAUGE_LOAD<UseTex, atype, Real>( arg.gaugefield, ids + newdir * DEVPARAMS::Volume, DEVPARAMS::size);
				ids = Index_4D_Neig_NM(ids, newdir, 1);			
			}
		}
	}
        
    msun tl = msun::identity();
    msun tr = msun::identity();
    for(int it = 0; it <= arg.Tmax; it++){

    int idt = Index_4D_Neig_NM(id, arg.nu, it);
    int idl = idt;
    int idr = Index_4D_Neig_NM(ids, arg.nu, it);
    
    msun linktop = msun::identity();    
	if(id < DEVPARAMS::Volume){
		for(int dir = 0; dir < 3; dir++){
			int newdir = dir;//( arg.mu + dir ) % 3;
			for(int radius = 0; radius < arg.radius[newdir]; radius++){
				linktop *= GAUGE_LOAD<UseTex, atype, Real>( arg.gaugefield, idt + newdir * DEVPARAMS::Volume, DEVPARAMS::size);
				idt = Index_4D_Neig_NM(idt, newdir, 1);			
			}
		}
	}
    
	complex wl = complex::zero();
	if(id < DEVPARAMS::Volume){
	  wl = (linkb * tr * linktop.dagger() * tl.dagger()).trace();
	}
	reduce_block_1d<complex>(arg.res + it, wl);


	if(id < DEVPARAMS::Volume){
		tl *= GAUGE_LOAD<UseTex, atype, Real>( arg.gaugefield_nosmear, idl + tdirvolume, DEVPARAMS::size);
		tr *= GAUGE_LOAD<UseTex, atype, Real>( arg.gaugefield_nosmear, idr + tdirvolume, DEVPARAMS::size);
	} 

  }
   // }
  //}
}





template <bool UseTex, class Real, ArrayType atype> 
class WilsonLoopRnoInt: Tunable{
private:
   WLArgRnoInt<Real> arg;
   gauge array;
   int size;
   double timesec;
#ifdef TIMMINGS
    Timer WilsonLooptime;
#endif
	TuneParam tp;

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
        tp = tuneLaunch(*this, getTuning(), getVerbosity());
        CUDA_SAFE_CALL(hipMemset(arg.res, 0, (arg.Tmax+1) * sizeof(complex)));
		kernel_WilsonLoopRnoInt<UseTex, Real, atype><<<tp.grid,tp.block, tp.block.x*sizeof(complex), stream>>>(arg);

}
public:
   WilsonLoopRnoInt(WLArgRnoInt<Real> arg, gauge array): arg(arg), array(array){
	size = 1;
	for(int i=0;i<4;i++){
		size *= PARAMS::Grid[i];
	} 
	timesec = 0.0;  
}
   ~WilsonLoopRnoInt(){};
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    WilsonLooptime.start();
#endif
    apply(stream);
    CUDA_SAFE_DEVICE_SYNC();
    CUT_CHECK_ERROR("Kernel execution failed");
#ifdef TIMMINGS
	CUDA_SAFE_DEVICE_SYNC( );
    WilsonLooptime.stop();
    timesec = WilsonLooptime.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const {	return 0;}
   long long bytes() const{return 0;}
   double time(){	return timesec;}
   void stat(){	COUT << "WilsonLoop:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size << ",prec="  << sizeof(Real);
    string tmp = "None";
    return TuneKey(vol.str().c_str(), typeid(*this).name(), tmp.c_str(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() {  }
  void postTune() {    }

};





template<bool UseTex, class Real>
void WilsonLoopR(gauge array, gauge array_nosmear, complex *res, int radius[3], int Tmax, int mu, int nu){
  Timer mtime;
  mtime.start(); 
  WLArgRnoInt<Real> arg;
	arg.gaugefield = array.GetPtr();
	arg.gaugefield_nosmear = array_nosmear.GetPtr();
	arg.res = (complex*) dev_malloc((Tmax+1)*sizeof(complex));
	for(int i = 0; i < 3; i++) arg.radius[i] = radius[i];	
	arg.Tmax = Tmax;
	arg.mu = mu;
	arg.nu = nu;

   
  
  if(array.Type() != SOA)
    errorCULQCD("Only defined for SOA arrays...\n");
  if(array.EvenOdd() == true)
    errorCULQCD("Not defined for EvenOdd arrays...\n");
  if(array_nosmear.Type() != SOA)
    errorCULQCD("Only defined for SOA arrays...\n");
  if(array_nosmear.EvenOdd() == true)
    errorCULQCD("Not defined for EvenOdd arrays...\n");


	WilsonLoopRnoInt<UseTex, Real, SOA> wl(arg, array);
	wl.Run();
	//wl.stat();


  CUDA_SAFE_CALL(hipMemcpy(res, arg.res, (arg.Tmax+1) * sizeof(complex), hipMemcpyDeviceToHost));
  dev_free(arg.res);
  for(int it = 0; it <= Tmax; it++)
    res[it] /= (Real)(PARAMS::Volume * NCOLORS);
  CUDA_SAFE_DEVICE_SYNC( );
  mtime.stop();
  COUT << "Time WilsonLoop:  " <<  mtime.getElapsedTimeInSec() << " s"  << endl;
}




template<class Real>
void WilsonLoopR(gauge array, gauge array_nosmear, complex *res, int radius[3], int Tmax, int mu, int nu){
  WilsonLoopR<false, Real>(array, array_nosmear, res, radius, Tmax, mu, nu);
}


template void WilsonLoopR<float>(gauges array, gauges array_nosmear, complexs *res, int radius[3], int Tmax, int mu, int nu);
template void WilsonLoopR<double>(gauged array, gauged array_nosmear, complexd *res, int radius[3], int Tmax, int mu, int nu);










}
