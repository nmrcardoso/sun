#include "hip/hip_runtime.h"


#include <stdio.h>
#include <string.h>
#include <iostream>
#include <iomanip>

#include <meas/plaquette.h>
#include <device_load_save.h>
#include <cuda_common.h>
#include <constants.h>
#include <index.h>
#include <reduction.h>
#include <timer.h>
#include <texture_host.h>
#include <comm_mpi.h>


#include <cudaAtomic.h>

#include <reduce_block_1d.h>

using namespace std;


namespace CULQCD{


//kernel to calculate the plaquette at each site of the lattice in EvenOdd order 
template <bool UseTex, ArrayType atype, class Real> 
__global__ void kernel_calc_plaquette_evenodd(complex *array, complex *plaquette ){
	uint idd = INDEX1D();
	
	
	complex plaq = complex::zero();
	if(idd < param_Volume()) {

		int oddbit = 0;
		int id = idd;
		if(idd >= param_HalfVolume()){
			oddbit = 1;
			id = idd - param_HalfVolume();
		}
		#ifdef MULTI_GPU
			int x[4];
			Index_4D_EO(x, id, oddbit);
			for(int i=0; i<4;i++) x[i] += param_border(i);
			int idxoddbit = ((((x[3] * param_GridG(2) + x[2]) * param_GridG(1)) + x[1] ) * param_GridG(0) + x[0]) >> 1 ;
			idxoddbit += oddbit  * param_HalfVolumeG();
			int mustride = DEVPARAMS::VolumeG;
			int offset = mustride * 4;
		#else
			int mustride = DEVPARAMS::Volume;
			int offset = mustride * 4;
			int idxoddbit = id + oddbit  * param_HalfVolume();
			//int idxoddbit = idd; //cuda reports error: misaligned address LOL

		#endif
		//------------------------------------------------------------------------
		// Calculate space-time plaquettes, stored in the real real of plaquette array
		//------------------------------------------------------------------------
		msun link, link1;
		//#pragma unroll
		for(int mu = 0; mu < 3; mu++){	
			link1 = GAUGE_LOAD<UseTex, atype,Real>( array, idxoddbit + mu * mustride, offset);
			int newidmu1 = Index_4D_Neig_EO(id, oddbit, mu, 1);
			//#pragma unroll
			for (int nu = (mu+1); nu < 4; nu++){
				link = GAUGE_LOAD<UseTex, atype,Real>( array,  newidmu1 + nu * mustride, offset);	      
				link *= GAUGE_LOAD_DAGGER<UseTex, atype,Real>( array, Index_4D_Neig_EO(id, oddbit, nu, 1) + mu * mustride, offset);			
				link *= GAUGE_LOAD_DAGGER<UseTex, atype,Real>( array, idxoddbit + nu * mustride, offset);
				if(nu == 3) plaq.imag() += (link1 * link).realtrace();
				else plaq.real() += (link1 * link).realtrace();
			}
		}
	}
    reduce_block_1d<complex>(plaquette, plaq);	  
}





template <class Real> 
Plaquette<Real>::Plaquette(gauge &array):array(array){
	plaq_value = complex::zero();
	size = 1;
	for(int i=0;i<4;i++){
		grid[i]=PARAMS::Grid[i];
		size *= PARAMS::Grid[i];
	} 
	timesec = 0.0;
    reduced = false;
    SetFunctionPtr();
	sum = (complex*) dev_malloc(sizeof(complex));
}
template <class Real> 
void Plaquette<Real>::SetFunctionPtr(){
	tex = PARAMS::UseTex;
	kernel_pointer = NULL;
	if(array.EvenOdd()){
	    if(tex){
			#if (NCOLORS == 3)
	        if(array.Type() == SOA) kernel_pointer = &kernel_calc_plaquette_evenodd<true, SOA, Real>;		
	        if(array.Type() == SOA12) kernel_pointer = &kernel_calc_plaquette_evenodd<true, SOA12, Real>;
	        if(array.Type() == SOA8) kernel_pointer = &kernel_calc_plaquette_evenodd<true, SOA8, Real>;
			#else
	        kernel_pointer = &kernel_calc_plaquette_evenodd<true, SOA, Real>;	
			#endif
	    }
	    else{
			#if (NCOLORS == 3)
	        if(array.Type() == SOA) kernel_pointer = &kernel_calc_plaquette_evenodd<false, SOA, Real>;
	        if(array.Type() == SOA12) kernel_pointer = &kernel_calc_plaquette_evenodd<false, SOA12, Real>;
	        if(array.Type() == SOA8) kernel_pointer = &kernel_calc_plaquette_evenodd<false, SOA8, Real>;
			#else
	        kernel_pointer = &kernel_calc_plaquette_evenodd<false, SOA, Real>;	
			#endif
	    }
	}
	if(kernel_pointer == NULL) errorCULQCD("No kernel plaquette function exist for this gauge array...");
}


template <class Real> 
void Plaquette<Real>::apply(const hipStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
 	  CUDA_SAFE_CALL(hipMemset(sum, 0, sizeof(complex)));
      kernel_pointer<<<tp.grid,tp.block, tp.block.x*sizeof(complex), stream>>>(array.GetPtr(), sum);
      reduced = false;
}
template <class Real> 
complex Plaquette<Real>::Run(const hipStream_t &stream){
#ifdef TIMMINGS
    plaqtime.start();
#endif
    //just ensure that the texture was not unbind somewhere...
    if(tex != PARAMS::UseTex){
    	SetFunctionPtr();
    } 
    GAUGE_TEXTURE(array.GetPtr(), true);
    apply(stream);
    CUDA_SAFE_CALL(hipMemcpy(&plaq_value, sum, sizeof(complex), hipMemcpyDeviceToHost));	
	plaq_value /= (Real)(3 * NCOLORS * size);
	#ifdef MULTI_GPU
	comm_Allreduce(&plaq_value);
	plaq_value /= numnodes();
	#endif
#ifdef TIMMINGS
	CUDA_SAFE_DEVICE_SYNC( );
    CUT_CHECK_ERROR("Kernel execution failed");
    plaqtime.stop();
    timesec = plaqtime.getElapsedTimeInSec();
#endif
	return plaq_value;
}
template <class Real> 
complex Plaquette<Real>::Run(){
	return Run(0);
}
template <class Real> 
complex Plaquette<Real>::Value() {
	return plaq_value;
}

template <class Real> 
double Plaquette<Real>::time(){
	return timesec;
}

template <class Real> 
void Plaquette<Real>::stat(){
	COUT << "Plaquette:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;
}


template <class Real> 
void Plaquette<Real>::printValue(){ 
	Real resf = (plaq_value.real() + plaq_value.imag()) / 2.0;
	//.real() space  and .imag() time part
	printfCULQCD("Plaquette: < %.12e : %.12e > :: mean: %.12e\n", plaq_value.real(), plaq_value.imag(),resf);
}



template <class Real> 
long long Plaquette<Real>::flop() const { 
    //NEED TO RECOUNT!!!!!! 
	#ifdef MULTI_GPU
	return (array.getNumFlop(true) + NCOLORS * NCOLORS * NCOLORS * 120LL) * size * numnodes();
	#else
	return NCOLORS * NCOLORS * NCOLORS * 120LL * size * numnodes();
	#endif
}
template <class Real> 
long long Plaquette<Real>::bytes() const {
    //NEED TO RECOUNT!!!!!!  
	#ifdef MULTI_GPU
	return (22LL * array.getNumParams() + 4LL) * size * numnodes() * sizeof(Real);
	#else
	return (22LL * array.getNumParams() + 4LL) * size * sizeof(Real);
	#endif
}

template <class Real> 
double Plaquette<Real>::flops(){
	return ((double)flop() * 1.0e-9) / timesec;
}
template <class Real> 
double Plaquette<Real>::bandwidth(){
	return (double)bytes() / (timesec * (double)(1 << 30));
}

template <class Real> 
Plaquette<Real>::~Plaquette(){
	dev_free(sum);
}


template class Plaquette<float>;
template class Plaquette<double>;




}
