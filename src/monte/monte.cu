#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <iostream>
#include <iomanip>

#ifdef MULTI_GPU
#include <mpi.h>
#endif

#include <timer.h>
#include <cuda_common.h>
#include <monte/monte.h>
#include <device_load_save.h>
#include <constants.h>
#include <matrixsun.h>
#include <gaugearray.h>
#include <index.h>
#include <device_PHB_OVR.h>
#include <reunitlink.h>
#include <staple.h>
#include <comm_mpi.h>
#include <exchange.h>
#include <texture_host.h>



#include "staple.cuh"


using namespace std;


namespace CULQCD{


/**
	@brief CUDA Kernel to perform pseudo-heatbath in a even/odd lattice order.
	@param array gauge field
	@param state CUDA RNG array state
	@param parity if 0 update even lattice sites, if 1 update odd lattice sites
	@param mu lattice direction to update links 
*/
template <bool UseTex, ArrayType atype, class Real, bool stapleSOA12type, int actiontype> 
__global__ void 
kernel_PHeatBath_evenodd(complex *array, complex *staple_array, cuRNGState *state, int oddbit, int mu){
	int id = INDEX1D();
	if(id >= param_HalfVolume()) return;	
	#ifdef MULTI_GPU
		int x[4];
		Index_4D_EO(x, id, oddbit);
		for(int i=0; i<4;i++)x[i]+=param_border(i);
		int idxoddbit = ((((x[3] * param_GridG(2) + x[2]) * param_GridG(1)) + x[1] ) * param_GridG(0) + x[0]) >> 1 ;
		idxoddbit += oddbit  * param_HalfVolumeG();
		int mustride = DEVPARAMS::VolumeG;
		int muvolume = mu * mustride;
		int offset = mustride * 4;
	#else
		int x[4];
		Index_4D_EO(x, id, oddbit);
		int idxoddbit = id + oddbit  * param_HalfVolume();
		int mustride = DEVPARAMS::Volume;
		int muvolume = mu * mustride;
		int offset = DEVPARAMS::size;
	#endif
	msun staple = msun::zero();
	if( actiontype == 1 || actiontype == 2 ){
		staple = GAUGE_LOAD<false, SOA, Real>( staple_array, id, param_HalfVolume());
	}
	else {
		if( stapleSOA12type )
			Staple_SOA12<UseTex, atype, Real>(array, mu, staple, x, id, oddbit, idxoddbit, mustride, muvolume, offset);
		else
			Staple<UseTex, atype, Real>(array, mu, staple, id, oddbit, idxoddbit, mustride, muvolume, offset);
	}
	//if(id==0) staple.print();
    cuRNGState localState = state[ id ];
    idxoddbit += muvolume;
	msun U = GAUGE_LOAD<UseTex, atype, Real>( array, idxoddbit, offset);
	heatBathSUN<Real>( U, staple.dagger(), localState );
    state[ id ] = localState;
	GAUGE_SAVE<atype, Real>( array, U, idxoddbit, offset);
}







template <class Real, int actiontype> 
HeatBath<Real, actiontype>::HeatBath(gauge &array, RNG &randstates):array(array), randstates(randstates){
	SetFunctionPtr();
	size = 1;
	for(int i=0;i<4;i++){
		grid[i]=PARAMS::Grid[i];
		size *= PARAMS::Grid[i];
	} 
	size = size >> 1;
	timesec = 0.0;
	if( actiontype == 1 || actiontype == 2 ) staple = GetStapleArray<Real>();
}

template <class Real, int actiontype> 
HeatBath<Real, actiontype>::~HeatBath(){ FreeStapleArray(); }


template <class Real, int actiontype> 
void HeatBath<Real, actiontype>::SetFunctionPtr(){
	kernel_pointer = NULL;
	tex = PARAMS::UseTex;
	if(array.EvenOdd()){
	    if(tex){
			#if (NCOLORS == 3)
	        if(array.Type() == SOA) kernel_pointer = &kernel_PHeatBath_evenodd<true, SOA, Real, false, actiontype>;		
	        if(array.Type() == SOA12) kernel_pointer = &kernel_PHeatBath_evenodd<true, SOA12, Real, true, actiontype>;
	        if(array.Type() == SOA8) kernel_pointer = &kernel_PHeatBath_evenodd<true, SOA8, Real, false, actiontype>;
			#else
	        kernel_pointer = &kernel_PHeatBath_evenodd<true, SOA, Real, false, actiontype>;	
			#endif
	    }
	    else{
			#if (NCOLORS == 3)
	        if(array.Type() == SOA) kernel_pointer = &kernel_PHeatBath_evenodd<false, SOA, Real, false, actiontype>;
	        if(array.Type() == SOA12) kernel_pointer = &kernel_PHeatBath_evenodd<false, SOA12, Real, true, actiontype>;
	        if(array.Type() == SOA8) kernel_pointer = &kernel_PHeatBath_evenodd<false, SOA8, Real, false, actiontype>;
			#else
	        kernel_pointer = &kernel_PHeatBath_evenodd<false, SOA, Real, false, actiontype>;	
			#endif
	    }
	}
	if(kernel_pointer == NULL) errorCULQCD("No kernel HeatBath function exist for this gauge array...");
}

template <class Real, int actiontype> 
void HeatBath<Real, actiontype>::apply(const hipStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if(actiontype == 1 || actiontype == 2) 
      	kernel_pointer<<<tp.grid, tp.block, 0, stream>>>(array.GetPtr(), staple->GetPtr(), randstates.state, parity, dir);
  	  else
      	kernel_pointer<<<tp.grid, tp.block, 0, stream>>>(array.GetPtr(), 0, randstates.state, parity, dir);
}
template <class Real, int actiontype> 
void HeatBath<Real, actiontype>::Run(const hipStream_t &stream){
#ifdef TIMMINGS
    mtime.start();
#endif
    //just ensure that the texture was not unbind somewhere...
    if(tex != PARAMS::UseTex){
    	SetFunctionPtr();
    } 
    GAUGE_TEXTURE(array.GetPtr(), true);
	for(parity=0; parity < 2; parity++)
	for(dir = 0; dir < 4; dir++){
		if(actiontype==1) CalculateStaple<Real>(array, parity, dir, 1);	
		else if(actiontype==2) CalculateStaple<Real>(array, parity, dir, 2);	    
		apply(stream);	
		//EXCHANGE DATA!!!!!
	    #ifdef MULTI_GPU
	    if(numnodes()>1){
			CUDA_SAFE_DEVICE_SYNC( );
			Exchange_gauge_border_links_gauge(array, dir, parity);
		}
	    #endif
	}
#ifdef TIMMINGS
	CUDA_SAFE_DEVICE_SYNC( );
    CUT_CHECK_ERROR("Kernel execution failed");
    mtime.stop();
    timesec = mtime.getElapsedTimeInSec();
#endif
}





template <class Real, int actiontype> 
void HeatBath<Real, actiontype>::Run(){
	Run(0);
}
template <class Real, int actiontype> 
double HeatBath<Real, actiontype>::time(){
	return timesec;
}

template <class Real, int actiontype> 
void HeatBath<Real, actiontype>::stat(){
	COUT << "HeatBath:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;
}


template <class Real, int actiontype> 
long long HeatBath<Real, actiontype>::flop() const {
    //minumum flop for heatbath
	//NEEEDDDDD TO RECOUNT THIS PART!!!!!!!!!!!!!!!!!!!!!!!!! 
	#if (NCOLORS == 3)
	long long stapleflop = 2268LL ;
	long long phbflop = 801LL;
	long long ThreadFlop_phb = (7LL * array.getNumFlop(true) + array.getNumFlop(false) + stapleflop + phbflop) * size;
	#else
	double phbflop = NCOLORS * NCOLORS * NCOLORS + (NCOLORS * ( NCOLORS - 1) / 2) * (46LL + 48LL+56LL * NCOLORS);
	double stapleflop = NCOLORS * NCOLORS * NCOLORS * 84LL ;
    long long ThreadFlop_phb = (stapleflop + phbflop) * size;
	#endif
	#ifdef MULTI_GPU
	return ThreadFlop_phb * numnodes();
	#else
	return ThreadFlop_phb;
	#endif
}
template <class Real, int actiontype> 
long long HeatBath<Real, actiontype>::bytes() const { 
    #ifdef MULTI_GPU
    return (20LL * array.getNumParams() * sizeof(Real) + 2LL * sizeof(cuRNGState)) * size * numnodes();
	#else
    return (20LL * array.getNumParams() * sizeof(Real) + 2LL * sizeof(cuRNGState))  * size;	
	#endif
}

template <class Real, int actiontype> 
double HeatBath<Real, actiontype>::flops(){
	return ((double)flop() * 8 * 1.0e-9) / timesec;
}
template <class Real, int actiontype> 
double HeatBath<Real, actiontype>::bandwidth(){
	return (double)bytes() * 8 / (timesec * (double)(1 << 30));
}






template class HeatBath<float, 0>;
template class HeatBath<float, 1>;
template class HeatBath<float, 2>;
template class HeatBath<double, 0>;
template class HeatBath<double, 1>;
template class HeatBath<double, 2>;

}
