#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <iostream>
#include <string>
#include <iomanip>
#include <typeinfo>
	


#include <gaugefix/gaugefix.h>
#include <cuda_common.h>
#include <comm_mpi.h>
#include <complex.h>
#include <matrixsun.h>

#include <tune.h>
#include <index.h>
#include <device_load_save.h>
#include <texture.h>
#include <texture_host.h>
#include <timer.h>
#include <reduction.h>
#include <constants.h>



namespace CULQCD{


#ifdef USE_GAUGE_FIX

template<int DIR, bool UseTex, ArrayType atype, class Real>
__global__ void  kernel_calc_Fg_theta_dir(complex *array, complex *res_save){
	int idd = INDEX1D();
	if(idd < DEVPARAMS::Volume){
		int id = idd;
		int oddbit = 0;
		if(idd >= DEVPARAMS::HalfVolume){
			id -= DEVPARAMS::HalfVolume;
			oddbit = 1;
		}
		int offset = DEVPARAMS::VolumeG * 4;
		int idx = EOIndeX(id, oddbit);
		msun delta = msun::zero();
		//Uplinks
		for(int nu = 0; nu < DIR; nu++) 
			delta -= GAUGE_LOAD<UseTex, atype, Real>( array,  idx + nu * DEVPARAMS::VolumeG, offset);
		complex res;
		//Fg (sum_DIR uplinks)
		res.real() = -delta.realtrace();
		//Downlinks
		for(int nu = 0; nu < DIR; nu++) 	
			delta += GAUGE_LOAD<UseTex, atype, Real>( array, neighborEOIndexMinusOne(id, oddbit, nu) + nu * DEVPARAMS::VolumeG, offset);
		delta = (delta-delta.dagger()).subtraceunit();
		//theta
		res.imag() = realtraceUVdagger(delta, delta);
		res_save[idd] = res;
	}
}


template<int DIR, bool UseTex, ArrayType atype, class Real>
void GaugeFixQuality<DIR, UseTex, atype, Real>::apply(const hipStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      kernel_calc_Fg_theta_dir<DIR, UseTex, atype, Real><<<tp.grid,tp.block, 0, stream>>>(array.GetPtr(), sum);
	}
template<int DIR, bool UseTex, ArrayType atype, class Real>
GaugeFixQuality<DIR, UseTex, atype, Real>::GaugeFixQuality(gauge &array):array(array){
	if(array.Type() != atype) errorCULQCD("gauge array type and template types do not match...");
	if(!array.EvenOdd()) errorCULQCD("gauge array must be a even/odd array...");
	functionName = "GaugeFixQuality";
	value = complex::zero();
	size = 1;
	for(int i=0;i<4;i++){
		grid[i]=PARAMS::Grid[i];
		size *= PARAMS::Grid[i];
	} 
	sum = (complex*)dev_malloc(size * sizeof(complex));
	timesec = 0.0;
}
template<int DIR, bool UseTex, ArrayType atype, class Real>
GaugeFixQuality<DIR, UseTex, atype, Real>::~GaugeFixQuality(){dev_free(sum);};
template<int DIR, bool UseTex, ArrayType atype, class Real>
complex GaugeFixQuality<DIR, UseTex, atype, Real>::Run(const hipStream_t &stream){
#ifdef TIMMINGS
    mtime.start();
#endif
    if(UseTex) BIND_GAUGE_TEXTURE(array.GetPtr());
    apply(stream);
	value = reduction<complex>(sum, size, stream);
	value /= (Real)(PARAMS::Volume * NCOLORS);
	value.real() /= (Real)DIR;
	#ifdef MULTI_GPU
	comm_Allreduce(&value);
	value /= numnodes();
	#endif
#ifdef TIMMINGS
	CUDA_SAFE_DEVICE_SYNC( );
    mtime.stop();
    timesec = mtime.getElapsedTimeInSec();
#endif
	return value;
}
template<int DIR, bool UseTex, ArrayType atype, class Real>
complex GaugeFixQuality<DIR, UseTex, atype, Real>::Run(){return Run(0);}
template<int DIR, bool UseTex, ArrayType atype, class Real>
double GaugeFixQuality<DIR, UseTex, atype, Real>::flops(){	return ((double)flop() * 1.0e-9) / timesec;}
template<int DIR, bool UseTex, ArrayType atype, class Real>
double GaugeFixQuality<DIR, UseTex, atype, Real>::bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
template<int DIR, bool UseTex, ArrayType atype, class Real>
long long GaugeFixQuality<DIR, UseTex, atype, Real>::flop() const { 
	long long arrayflops = 2LL * DIR * array.getNumFlop(true);
	return (arrayflops + 2LL * NCOLORS * NCOLORS * (DIR + 1) + 4LL * NCOLORS * ( 1 + NCOLORS) ) * size * numnodes();
}
template<int DIR, bool UseTex, ArrayType atype, class Real>
long long GaugeFixQuality<DIR, UseTex, atype, Real>::bytes() const { 
	return (2LL * DIR * array.getNumParams() + 2LL) * size * sizeof(Real) * numnodes();
}

template<int DIR, bool UseTex, ArrayType atype, class Real>
double GaugeFixQuality<DIR, UseTex, atype, Real>::time(){return timesec;}

template<int DIR, bool UseTex, ArrayType atype, class Real>
void GaugeFixQuality<DIR, UseTex, atype, Real>::stat(){
COUT << "GaugeFixQuality:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;
}
template<int DIR, bool UseTex, ArrayType atype, class Real>
void GaugeFixQuality<DIR, UseTex, atype, Real>::printValue(){
	printfCULQCD("GaugeFixQuality:Fg = %.12e\ttheta = %.12e\n", value.real(), value.imag() );
}




#if (NCOLORS == 3)
template class GaugeFixQuality<4, true, SOA, float>;
template class GaugeFixQuality<4, true, SOA12, float>;
template class GaugeFixQuality<4, true, SOA8, float>;

template class GaugeFixQuality<4, true, SOA, double>;
template class GaugeFixQuality<4, true, SOA12, double>;
template class GaugeFixQuality<4, true, SOA8, double>;


template class GaugeFixQuality<4, false, SOA, float>;
template class GaugeFixQuality<4, false, SOA12, float>;
template class GaugeFixQuality<4, false, SOA8, float>;

template class GaugeFixQuality<4, false, SOA, double>;
template class GaugeFixQuality<4, false, SOA12, double>;
template class GaugeFixQuality<4, false, SOA8, double>;


template class GaugeFixQuality<3, true, SOA, float>;
template class GaugeFixQuality<3, true, SOA12, float>;
template class GaugeFixQuality<3, true, SOA8, float>;

template class GaugeFixQuality<3, true, SOA, double>;
template class GaugeFixQuality<3, true, SOA12, double>;
template class GaugeFixQuality<3, true, SOA8, double>;


template class GaugeFixQuality<3, false, SOA, float>;
template class GaugeFixQuality<3, false, SOA12, float>;
template class GaugeFixQuality<3, false, SOA8, float>;

template class GaugeFixQuality<3, false, SOA, double>;
template class GaugeFixQuality<3, false, SOA12, double>;
template class GaugeFixQuality<3, false, SOA8, double>;
#elif (NCOLORS > 3)
template class GaugeFixQuality<4, true, SOA, float>;
template class GaugeFixQuality<4, true, SOA, double>;

template class GaugeFixQuality<4, false, SOA, float>;
template class GaugeFixQuality<4, false, SOA, double>;


template class GaugeFixQuality<3, true, SOA, float>;
template class GaugeFixQuality<3, true, SOA, double>;

template class GaugeFixQuality<3, false, SOA, float>;
template class GaugeFixQuality<3, false, SOA, double>;
#else
#error Code not done for NCOLORS < 3
#endif






#endif








}
